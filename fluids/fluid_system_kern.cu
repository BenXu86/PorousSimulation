
#include <hip/hip_runtime.h>
/*
  FLUIDS v.3 - SPH Fluid Simulator for CPU and GPU
  Copyright (C) 2012. Rama Hoetzlein, http://fluids3.com

  Fluids-ZLib license (* see part 1 below)
  This software is provided 'as-is', without any express or implied
  warranty.  In no event will the authors be held liable for any damages
  arising from the use of this software.

  Permission is granted to anyone to use this software for any purpose,
  including commercial applications, and to alter it and redistribute it
  freely, subject to the following restrictions:

  1. The origin of this software must not be misrepresented; you must not
     claim that you wrote the original software. Acknowledgement of the
	 original author is required if you publish this in a paper, or use it
	 in a product. (See fluids3.com for details)
  2. Altered source versions must be plainly marked as such, and must not be
     misrepresented as being the original software.
  3. This notice may not be removed or altered from any source distribution.
*/
