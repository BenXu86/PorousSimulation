/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

#ifdef _WIN32
#  define NOMINMAX 
#endif

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cutil_inline.h>

// includes, kernels
#include <scan.cu>  // defines prescanArray()

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

// regression test functionality
extern "C" 
unsigned int compare( const float* reference, const float* data, 
                     const unsigned int len);
extern "C" 
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a scan test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

#ifndef __DEVICE_EMULATION__
    unsigned int num_test_iterations = 100;
    unsigned int num_elements = 1000000; // can support large, non-power-of-2 arrays!
#else
    unsigned int num_test_iterations = 1;
    unsigned int num_elements = 10000; // can support large, non-power-of-2 arrays!
#endif
    
    cutGetCmdLineArgumenti( argc, (const char**) argv, "n", (int*)&num_elements);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "i", (int*)&num_test_iterations);

    unsigned int mem_size = sizeof( float) * num_elements;
    
    unsigned int timerGPU, timerCPU;
    cutilCheckError(cutCreateTimer(&timerCPU));
    cutilCheckError(cutCreateTimer(&timerGPU));

    // allocate host memory to store the input data
    float* h_data = (float*) malloc( mem_size);
      
    // initialize the input data on the host
    for( unsigned int i = 0; i < num_elements; ++i) 
    {
        h_data[i] = 1.0f;//(int)(10 * rand()/32768.f);
    }

    // compute reference solution
    float* reference = (float*) malloc( mem_size); 
    cutStartTimer(timerCPU);
    for (unsigned int i = 0; i < num_test_iterations; i++)
    {
        computeGold( reference, h_data, num_elements);
    }
    cutStopTimer(timerCPU);

    // allocate device memory input and output arrays
    float* d_idata = NULL;
    float* d_odata = NULL;

    cutilSafeCall( hipMalloc( (void**) &d_idata, mem_size));
    cutilSafeCall( hipMalloc( (void**) &d_odata, mem_size));
    
    // copy host memory to device input array
    cutilSafeCall( hipMemcpy( d_idata, h_data, mem_size, hipMemcpyHostToDevice) );
    // initialize all the other device arrays to be safe
    cutilSafeCall( hipMemcpy( d_odata, h_data, mem_size, hipMemcpyHostToDevice) );

    printf("Running parallel prefix sum (prescan) of %d elements\n", num_elements);
    printf("This version is work efficient (O(n) adds)\n");
    printf("and has very few shared memory bank conflicts\n\n");

    preallocBlockSums(num_elements);

    // run once to remove startup overhead
    prescanArray(d_odata, d_idata, num_elements);


    // Run the prescan
    cutStartTimer(timerGPU);
    for (unsigned int i = 0; i < num_test_iterations; i++)
    {
        //printf("prescanArray\n");
        prescanArray(d_odata, d_idata, num_elements);
    }
    cutStopTimer(timerGPU);

    deallocBlockSums();    

    // copy result from device to host
    cutilSafeCall(hipMemcpy( h_data, d_odata, sizeof(float) * num_elements, 
                               hipMemcpyDeviceToHost));

    // If this is a regression test write the results to a file
    if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression")) 
    {
        // write file for regression test 
        cutWriteFilef( "./data/result.dat", h_data, num_elements, 0.0);
    }
    else 
    {
        // custom output handling when no regression test running
        // in this case check if the result is equivalent to the expected soluion
        unsigned int result_regtest = cutComparef( reference, h_data, num_elements);
        printf( "Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");
        printf( "Average GPU execution time: %f ms\n", cutGetTimerValue(timerGPU) / num_test_iterations);
        printf( "CPU execution time:         %f ms\n", cutGetTimerValue(timerCPU) / num_test_iterations);
    }

    printf("\nCheck out the CUDA Data Parallel Primitives Library for more on scan.\n");
    printf("http://www.gpgpu.org/developer/cudpp\n");

    // cleanup memory
    cutDeleteTimer(timerCPU);
    cutDeleteTimer(timerGPU);
    free( h_data);
    free( reference);
    hipFree( d_odata);
    hipFree( d_idata);

    hipDeviceReset();
}
