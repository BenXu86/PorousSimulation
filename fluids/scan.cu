#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <scanLargeArray_kernel.cu>
#include <assert.h>

inline bool 
isPowerOfTwo(int n)
{
    return ((n&(n-1))==0) ;
}

inline int 
floorPow2(int n)
{
#ifdef WIN32
    // method 2
    return 1 << (int)logb((float)n);
#else
    // method 1
    // float nf = (float)n;
    // return 1 << (((*(int*)&nf) >> 23) - 127); 
    int exp;
    frexp((float)n, &exp);
    return 1 << (exp - 1);
#endif
}

#define BLOCK_SIZE 256

float** g_scanBlockSums;
unsigned int g_numEltsAllocated = 0;
unsigned int g_numLevelsAllocated = 0;

void preallocBlockSums(unsigned int maxNumElements)
{
    assert(g_numEltsAllocated == 0); // shouldn't be called 

    g_numEltsAllocated = maxNumElements;

    unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
    unsigned int numElts = maxNumElements;

    int level = 0;

    do
    {       
        unsigned int numBlocks = 
            max(1, (int)ceil((float)numElts / (2.f * blockSize)));
        if (numBlocks > 1)
        {
            level++;
        }
        numElts = numBlocks;
    } while (numElts > 1);

    g_scanBlockSums = (float**) malloc(level * sizeof(float*));
    g_numLevelsAllocated = level;
    
    numElts = maxNumElements;
    level = 0;
    
    do
    {       
        unsigned int numBlocks = 
            max(1, (int)ceil((float)numElts / (2.f * blockSize)));
        if (numBlocks > 1) 
        {
            cutilSafeCall(hipMalloc((void**) &g_scanBlockSums[level++],  
                                      numBlocks * sizeof(float)));
        }
        numElts = numBlocks;
    } while (numElts > 1);

    cutilCheckMsg("preallocBlockSums");
}

void deallocBlockSums()
{
    for (unsigned int i = 0; i < g_numLevelsAllocated; i++)
    {
        hipFree(g_scanBlockSums[i]);
    }

    cutilCheckMsg("deallocBlockSums");
    
    free((void**)g_scanBlockSums);

    g_scanBlockSums = 0;
    g_numEltsAllocated = 0;
    g_numLevelsAllocated = 0;
}


void prescanArrayRecursive(float *outArray, 
                           const float *inArray, 
                           int numElements, 
                           int level)
{
    unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
    unsigned int numBlocks = 
        max(1, (int)ceil((float)numElements / (2.f * blockSize)));
    unsigned int numThreads;

    if (numBlocks > 1)
        numThreads = blockSize;
    else if (isPowerOfTwo(numElements))
        numThreads = numElements / 2;
    else
        numThreads = floorPow2(numElements);

    unsigned int numEltsPerBlock = numThreads * 2;

    // if this is a non-power-of-2 array, the last block will be non-full
    // compute the smallest power of 2 able to compute its scan.
    unsigned int numEltsLastBlock = 
        numElements - (numBlocks-1) * numEltsPerBlock;
    unsigned int numThreadsLastBlock = max(1, numEltsLastBlock / 2);
    unsigned int np2LastBlock = 0;
    unsigned int sharedMemLastBlock = 0;
    
    if (numEltsLastBlock != numEltsPerBlock)
    {
        np2LastBlock = 1;

        if(!isPowerOfTwo(numEltsLastBlock))
            numThreadsLastBlock = floorPow2(numEltsLastBlock);    
        
        unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
        sharedMemLastBlock = 
            sizeof(float) * (2 * numThreadsLastBlock + extraSpace);
    }

    // padding space is used to avoid shared memory bank conflicts
    unsigned int extraSpace = numEltsPerBlock / NUM_BANKS;
    unsigned int sharedMemSize = 
        sizeof(float) * (numEltsPerBlock + extraSpace);

#ifdef DEBUG
    if (numBlocks > 1)
    {
        assert(g_numEltsAllocated >= numElements);
    }
#endif

    // setup execution parameters
    // if NP2, we process the last block separately
    dim3  grid(max(1, numBlocks - np2LastBlock), 1, 1); 
    dim3  threads(numThreads, 1, 1);

    // make sure there are no CUDA errors before we start
    cutilCheckMsg("prescanArrayRecursive before kernels");

    // execute the scan
    if (numBlocks > 1)
    {
        prescan<true, false><<< grid, threads, sharedMemSize >>>(outArray, 
                                                                 inArray, 
                                                                 g_scanBlockSums[level],
                                                                 numThreads * 2, 0, 0);
        cutilCheckMsg("prescanWithBlockSums");
        if (np2LastBlock)
        {
            prescan<true, true><<< 1, numThreadsLastBlock, sharedMemLastBlock >>>
                (outArray, inArray, g_scanBlockSums[level], numEltsLastBlock, 
                 numBlocks - 1, numElements - numEltsLastBlock);
            cutilCheckMsg("prescanNP2WithBlockSums");
        }

        // After scanning all the sub-blocks, we are mostly done.  But now we 
        // need to take all of the last values of the sub-blocks and scan those.  
        // This will give us a new value that must be sdded to each block to 
        // get the final results.
        // recursive (CPU) call
        prescanArrayRecursive(g_scanBlockSums[level], 
                              g_scanBlockSums[level], 
                              numBlocks, 
                              level+1);

        uniformAdd<<< grid, threads >>>(outArray, 
                                        g_scanBlockSums[level], 
                                        numElements - numEltsLastBlock, 
                                        0, 0);
        cutilCheckMsg("uniformAdd");
        if (np2LastBlock)
        {
            uniformAdd<<< 1, numThreadsLastBlock >>>(outArray, 
                                                     g_scanBlockSums[level], 
                                                     numEltsLastBlock, 
                                                     numBlocks - 1, 
                                                     numElements - numEltsLastBlock);
            cutilCheckMsg("uniformAdd");
        }
    }
    else if (isPowerOfTwo(numElements))
    {
        prescan<false, false><<< grid, threads, sharedMemSize >>>(outArray, inArray,
                                                                  0, numThreads * 2, 0, 0);
        cutilCheckMsg("prescan");
    }
    else
    {
         prescan<false, true><<< grid, threads, sharedMemSize >>>(outArray, inArray, 
                                                                  0, numElements, 0, 0);
         cutilCheckMsg("prescanNP2");
    }
}

void prescanArray(float *outArray, float *inArray, int numElements)
{
    prescanArrayRecursive(outArray, inArray, numElements, 0);
}


#endif // _PRESCAN_CU_
